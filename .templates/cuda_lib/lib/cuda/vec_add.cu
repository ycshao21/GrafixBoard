#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include "_template_project_name_/math/vec_add.hpp"
#include "_template_project_name_/utils/address.hpp"

namespace _template_project_name_::cuda
{

__global__ void vec_add(const float* a, const float* b, float* c, int n)
{
    const std::uint32_t threadIndex = threadIdx.x;
    std::uint32_t smId;
    asm volatile("mov.u32 %0, %smid;" : "=r"(smId));
    std::uint32_t warpId;
    asm volatile("mov.u32 %0, %warpid;" : "=r"(warpId));
    std::uint32_t laneId;
    asm volatile("mov.u32 %0, %laneid;" : "=r"(laneId));

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        ::printf("Thread %d: %f + %f = %f\n", i, a[i], b[i], a[i] + b[i]);
        c[i] = a[i] + b[i];
        auto offset = computeOffset<std::uint32_t>(1, 2, 3, 4, 5, 6);
        ::printf("Offset: %d\n", offset);
        ::printf("SM: %d | Warp: %d | Lane: %d | Thread %d - Here!\n", smId,
                 warpId, laneId, threadIndex);
    }
}

void launch_vec_add(const float* const a, const float* const b, float* const c,
                    const int n)
{
    ::printf("Hello World from CUDA!\n");
    ::printf("Vector size: %d\n", n);
    std::uint32_t block_size = 256;
    std::uint32_t grid_size = (n + block_size - 1) / block_size;

    // Apply Device Memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, n * sizeof(float));
    hipMalloc((void**) &d_b, n * sizeof(float));
    hipMalloc((void**) &d_c, n * sizeof(float));

    // Copy Host Memory to Device Memory
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch Kernel
    vec_add<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

    // Copy Device Memory to Host Memory
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free Device Memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

}  // namespace _template_project_name_::cuda